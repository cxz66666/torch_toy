#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <random>
#include <chrono>
#include <hip/hip_runtime_api.h>
#include <fstream>
#include <cstdint>
#include <hip/hip_bf16.h>
#include <hip/hip_fp16.h>
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/memcpy_async.h>

// ==========================================================================
// 1. 缺失的定义补充
// ==========================================================================

/**
 * @brief 计算向上取整的整数除法 (ceil(a/b))。
 * @param a 分子
 * @param b 分母
 */
#define ITER(x, y) (x + y - 1) / y

template <typename DATA_TYPE,
          typename std::enable_if_t<
              !std::is_same<__half, DATA_TYPE>::value &&
              !std::is_same<__hip_bfloat16, DATA_TYPE>::value> * = nullptr>
__device__ __forceinline__ void fastSpecializedAtomicAdd(DATA_TYPE *base,
                                                         int64_t offset,
                                                         const int64_t length,
                                                         DATA_TYPE value) {
  atomicAdd(base + offset, value);
}

template <class DATA_TYPE>
__device__ __forceinline__ void fastAtomicAdd(DATA_TYPE *base, int64_t offset,
                                              const int64_t length,
                                              DATA_TYPE value) {
  fastSpecializedAtomicAdd(base, offset, length, value);
}

// struct SharedMemory {
//   int edge_in_tile[TILE_K_PER_BLOCK];
//   int edge_out_tile[TILE_K_PER_BLOCK];
//   // 双缓冲，用于隐藏内存延迟
//   // 布局: [buffer_idx][warp_idx][element_idx]
//   DATA_TYPE emb_buffer[2][NUM_WARPS * EMB_DIM_STATIC];
// };

template <typename DATA_TYPE, int TILE_K_PER_BLOCK, int BLOCK_THREADS>
__global__ void __launch_bounds__(BLOCK_THREADS)
    gpu_pooling_forward_async_kernel(const DATA_TYPE *__restrict__ emb_table,
                                     const int *__restrict__ edge_in,
                                     const int *__restrict__ edge_out,
                                     DATA_TYPE *__restrict__ pooling_table,
                                     const int64_t emb_dim,
                                     const int edge_length) {
  constexpr int WARP_SIZE = 32;
  constexpr int NUM_WARPS = BLOCK_THREADS / WARP_SIZE;
  const int thread_id = threadIdx.x;
  const int warp_id = thread_id / WARP_SIZE;
  const int lane_id = thread_id % WARP_SIZE;

  // 动态共享内存布局
  extern __shared__ char smem_storage[];

  // 1. 为 edge_in/edge_out 分配空间
  int *smem_edge_in_tile = reinterpret_cast<int *>(smem_storage);
  int *smem_edge_out_tile = smem_edge_in_tile + TILE_K_PER_BLOCK;

  // 2. 为 warp 的循环变量分配空间
  int *smem_k_base = smem_edge_out_tile + TILE_K_PER_BLOCK;

  // 3. 为指针偏移量分配空间
  int *smem_ptr_offsets = smem_k_base + NUM_WARPS;

  // 计算 Block 处理的边范围
  const int block_tile_start = blockIdx.x * TILE_K_PER_BLOCK;
  if (block_tile_start >= edge_length) return;
  const int block_tile_end =
      min(block_tile_start + TILE_K_PER_BLOCK, edge_length);
  const int block_tile_size = block_tile_end - block_tile_start;

  // 预加载 edge 索引
  for (int i = thread_id; i < block_tile_size; i += BLOCK_THREADS) {
    smem_edge_in_tile[i] = edge_in[block_tile_start + i];
    smem_edge_out_tile[i] = edge_out[block_tile_start + i];
  }
  __syncthreads();

  // 初始化 warp 的循环变量
  if (thread_id < NUM_WARPS) {
    smem_k_base[thread_id] = 0;
  }
  __syncthreads();

  // 主循环
  while (true) {
    int k_base = -1;
    if (warp_id < NUM_WARPS) {
      k_base = smem_k_base[warp_id];
      smem_k_base[warp_id] += NUM_WARPS;  // 下次处理的起始位置
    }

    if (k_base >= block_tile_size) break;

    const int k_warp = k_base + warp_id;
    if (k_warp >= block_tile_size) continue;

    // 计算指针偏移量并存储到共享内存
    const int in_offset = smem_edge_in_tile[k_warp] * emb_dim;
    const int out_offset = smem_edge_out_tile[k_warp] * emb_dim;
    if (lane_id == 0) {
      smem_ptr_offsets[warp_id * 2] = in_offset;
      smem_ptr_offsets[warp_id * 2 + 1] = out_offset;
    }
    __syncwarp();

    // 从共享内存加载指针偏移
    const int in_off = smem_ptr_offsets[warp_id * 2];
    const int out_off = smem_ptr_offsets[warp_id * 2 + 1];
    const DATA_TYPE *gmem_src_ptr = emb_table + in_off;
    DATA_TYPE *gmem_dst_ptr = pooling_table + out_off;

    // 处理embedding维度
    for (int j = lane_id; j < emb_dim; j += WARP_SIZE) {
      atomicAdd(&gmem_dst_ptr[j], gmem_src_ptr[j]);
    }
  }
}
// 宏定义用于检查CUDA API调用的返回状态
#define CUDA_CHECK(call)                                               \
  do {                                                                 \
    hipError_t status = call;                                         \
    if (status != hipSuccess) {                                       \
      fprintf(stderr, "CUDA Error at %s:%d: %s\n", __FILE__, __LINE__, \
              hipGetErrorString(status));                             \
      exit(EXIT_FAILURE);                                              \
    }                                                                  \
  } while (0)

// ==========================================================================
// 3. Main函数：用于数据构造、内核调用和性能测试
// ==========================================================================

int main() {
  // ---- 1. 参数定义 ----
  // (此部分保持不变)

  using DataType = float;
  const int TILE_INDICES_VAL = 512;
  const int BLOCK_SIZE = 512;
  std::ifstream inFile("binary_data.bin", std::ios::binary);

  int edge_length, emb_table_length, pooling_table_length;
  int64_t emb_dim;
  inFile.read(reinterpret_cast<char *>(&edge_length), sizeof(int));
  inFile.read(reinterpret_cast<char *>(&emb_table_length), sizeof(int));
  inFile.read(reinterpret_cast<char *>(&pooling_table_length), sizeof(int));
  inFile.read(reinterpret_cast<char *>(&emb_dim), sizeof(int64_t));

  int *edge_in_cpu = reinterpret_cast<int *>(malloc(edge_length * sizeof(int)));
  int *edge_out_cpu =
      reinterpret_cast<int *>(malloc(edge_length * sizeof(int)));
  DataType *emb_table_cpu =
      reinterpret_cast<DataType *>(malloc(emb_table_length * sizeof(DataType)));
  DataType *pooling_table_cpu = reinterpret_cast<DataType *>(
      malloc(pooling_table_length * sizeof(DataType)));
  inFile.read(reinterpret_cast<char *>(edge_in_cpu), edge_length * sizeof(int));
  inFile.read(reinterpret_cast<char *>(edge_out_cpu),
              edge_length * sizeof(int));
  inFile.read(reinterpret_cast<char *>(emb_table_cpu),
              emb_table_length * sizeof(float));

  std::cout << "===== KERNEL PERFORMANCE TEST (CORRECTED) =====" << std::endl;
  std::cout << "Data Type: float" << std::endl;
  std::cout << "Embedding Dim: " << emb_dim << std::endl;
  std::cout << "Edge Length: " << edge_length << std::endl;
  std::cout << "===============================================" << std::endl;

  DataType *d_emb_table, *d_pooling_table;
  int *d_edge_in, *d_edge_out;

  CUDA_CHECK(hipMalloc(&d_emb_table, emb_table_length * sizeof(DataType)));
  CUDA_CHECK(hipMalloc(&d_edge_in, edge_length * sizeof(int)));
  CUDA_CHECK(hipMalloc(&d_edge_out, edge_length * sizeof(int)));
  CUDA_CHECK(
      hipMalloc(&d_pooling_table, pooling_table_length * sizeof(DataType)));

  CUDA_CHECK(hipMemcpy(d_emb_table, emb_table_cpu,
                        emb_table_length * sizeof(DataType),
                        hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_edge_in, edge_in_cpu, edge_length * sizeof(int),
                        hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_edge_out, edge_out_cpu, edge_length * sizeof(int),
                        hipMemcpyHostToDevice));
  CUDA_CHECK(
      hipMemset(d_pooling_table, 0, pooling_table_length * sizeof(DataType)));

  const dim3 blockDim(BLOCK_SIZE);
  const dim3 gridDim(ITER(edge_length, TILE_INDICES_VAL));
  std::cout << "Grid Dim: " << gridDim.x << ", Block Dim: " << blockDim.x
            << std::endl;
  size_t smem_size =
      2 * TILE_INDICES_VAL * sizeof(int) + 3 * (BLOCK_SIZE / 32) * sizeof(int);

  hipEvent_t start, stop;
  CUDA_CHECK(hipEventCreate(&start));
  CUDA_CHECK(hipEventCreate(&stop));

  gpu_pooling_forward_async_kernel<DataType, TILE_INDICES_VAL, BLOCK_SIZE>
      <<<gridDim, blockDim, smem_size>>>(d_emb_table, d_edge_in, d_edge_out,
                                         d_pooling_table, emb_dim, edge_length);

  CUDA_CHECK(hipDeviceSynchronize());

  // int num_runs = 100;

  // CUDA_CHECK(hipEventRecord(start));
  // for (int i = 0; i < num_runs; ++i) {
  //   gpu_pooling_forward_async_kernel<DataType, TILE_INDICES_VAL, BLOCK_SIZE>
  //       <<<gridDim, blockDim, smem_size>>>(d_emb_table, d_edge_in,
  //       d_edge_out,
  //                                          d_pooling_table, emb_dim,
  //                                          edge_length);
  // }
  // CUDA_CHECK(hipEventRecord(stop));

  // CUDA_CHECK(hipEventSynchronize(stop));
  // float total_time = 0;
  // CUDA_CHECK(hipEventElapsedTime(&total_time, start, stop));

  // float average_time_ms = total_time / num_runs;
  // std::cout << "\n--- Performance Results ---" << std::endl;
  // std::cout << "Number of test runs: " << num_runs << std::endl;
  // std::cout << "Average kernel execution time: " << average_time_ms << " ms"
  //           << std::endl;

  // std::cout << "\n--- Verification ---" << std::endl;

  // std::cout
  //     << "Resetting GPU buffer and running kernel once for verification..."
  //     << std::endl;
  // CUDA_CHECK(
  //     hipMemset(d_pooling_table, 0, pooling_table_length *
  //     sizeof(DataType)));

  // gpu_pooling_forward_async_kernel<DataType, TILE_INDICES_VAL, BLOCK_SIZE>
  //     <<<gridDim, blockDim, smem_size>>>(d_emb_table, d_edge_in, d_edge_out,
  //                                        d_pooling_table, emb_dim,
  //                                        edge_length);
  // CUDA_CHECK(hipDeviceSynchronize());  // 确保内核执行完毕

  // std::vector<DataType> h_gpu_result(pooling_table_length);
  // CUDA_CHECK(hipMemcpy(h_gpu_result.data(), d_pooling_table,
  //                       h_gpu_result.size() * sizeof(DataType),
  //                       hipMemcpyDeviceToHost));
  // memset(pooling_table_cpu, 0, pooling_table_length * sizeof(DataType));

  // std::cout << "Calculating reference result on CPU..." << std::endl;
  // for (int i = 0; i < edge_length; ++i) {
  //   int in_node = edge_in_cpu[i];
  //   int out_node = edge_out_cpu[i];
  //   for (int64_t d = 0; d < emb_dim; ++d) {
  //     pooling_table_cpu[out_node * emb_dim + d] +=
  //         emb_table_cpu[in_node * emb_dim + d];
  //   }
  // }

  // double total_absolute_error = 0.0;
  // for (size_t i = 0; i < pooling_table_length; ++i) {
  //   total_absolute_error += std::abs(pooling_table_cpu[i] - h_gpu_result[i]);
  // }

  // std::cout << "Total absolute error between CPU and GPU: "
  //           << total_absolute_error << std::endl;
  // if (total_absolute_error < 1e-1) {
  //   std::cout << "Result verification PASSED." << std::endl;
  // } else {
  //   std::cout << "Result verification FAILED." << std::endl;
  // }

  CUDA_CHECK(hipEventDestroy(start));
  CUDA_CHECK(hipEventDestroy(stop));
  CUDA_CHECK(hipFree(d_emb_table));
  CUDA_CHECK(hipFree(d_edge_in));
  CUDA_CHECK(hipFree(d_edge_out));
  CUDA_CHECK(hipFree(d_pooling_table));

  return 0;
}
