#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <random>
#include <chrono>
#include <hip/hip_runtime_api.h>
#include <fstream>
#include <cstdint>
#include <hip/hip_bf16.h>
#include <hip/hip_fp16.h>
#include <hip/hip_cooperative_groups.h>
#include <cuda/barrier>
#include <cooperative_groups/memcpy_async.h>
namespace cg = cooperative_groups;

// ==========================================================================
// 1. 缺失的定义补充
// ==========================================================================

/**
 * @brief 计算向上取整的整数除法 (ceil(a/b))。
 * @param a 分子
 * @param b 分母
 */
#define ITER(x, y) (x + y - 1) / y

template <typename DATA_TYPE,
          typename std::enable_if_t<
              !std::is_same<__half, DATA_TYPE>::value &&
              !std::is_same<__hip_bfloat16, DATA_TYPE>::value> * = nullptr>
__device__ __forceinline__ void fastSpecializedAtomicAdd(DATA_TYPE *base,
                                                         int64_t offset,
                                                         const int64_t length,
                                                         DATA_TYPE value) {
  atomicAdd(base + offset, value);
}

template <class DATA_TYPE>
__device__ __forceinline__ void fastAtomicAdd(DATA_TYPE *base, int64_t offset,
                                              const int64_t length,
                                              DATA_TYPE value) {
  fastSpecializedAtomicAdd(base, offset, length, value);
}

__global__ void mark_counts_cas_kernel(const int* edge_out, int* visit_num, int n) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = gridDim.x * blockDim.x;

  for (; i < n; i += stride) {
      int val = edge_out[i];
      int* addr = &visit_num[val];
      atomicAdd(addr, 1);
  }
}


template <typename DATA_TYPE, int TILE_K_PER_BLOCK, int BLOCK_THREADS>
__global__ void __launch_bounds__(BLOCK_THREADS)
    gpu_pooling_forward_async_kernel(const DATA_TYPE *__restrict__ emb_table,
                                     const int *__restrict__ edge_in,
                                     const int *__restrict__ edge_out,
                                     DATA_TYPE *__restrict__ pooling_table,
                                     const int64_t emb_dim,
                                     const int edge_length,
                                     const int *__restrict__ visit_num) {
  constexpr int WARP_SIZE = 32;
  constexpr int NUM_WARPS = BLOCK_THREADS / WARP_SIZE;
  const int thread_id = threadIdx.x;
  const int warp_id = thread_id / WARP_SIZE;
  const int lane_id = thread_id % WARP_SIZE;

  // 动态共享内存布局
  extern __shared__ char smem_storage[];

  // 1. 边索引存储
  int *smem_edge_in = reinterpret_cast<int *>(smem_storage);
  int *smem_edge_out = smem_edge_in + TILE_K_PER_BLOCK;
  int *smem_visit_num = smem_edge_out + TILE_K_PER_BLOCK;
  DATA_TYPE *smem_emb_buffer =
      reinterpret_cast<DATA_TYPE *>(smem_edge_out + TILE_K_PER_BLOCK);
  // 计算块处理的边范围
  const int block_tile_start = blockIdx.x * TILE_K_PER_BLOCK;
  if (block_tile_start >= edge_length) return;
  const int block_tile_end =
      min(block_tile_start + TILE_K_PER_BLOCK, edge_length);
  const int block_tile_size = block_tile_end - block_tile_start;

  // 预加载边索引
  for (int i = thread_id; i < block_tile_size; i += BLOCK_THREADS) {
    smem_edge_in[i] = edge_in[block_tile_start + i];
    smem_edge_out[i] = edge_out[block_tile_start + i];
    smem_visit_num[i] = visit_num[smem_edge_out[i]];
  }
  int k_base = warp_id * (TILE_K_PER_BLOCK / NUM_WARPS);
  __syncthreads();

  auto warp = cg::tiled_partition<WARP_SIZE>(cg::this_thread_block());
 
  int last_offset = -1;
  DATA_TYPE *shard_in_ptr = smem_emb_buffer + warp_id * emb_dim;

  // 主处理循环
  while (true) {
    // 检查是否完成
    if (k_base >= (warp_id + 1) * TILE_K_PER_BLOCK / NUM_WARPS) break;

    // 每个warp处理一个边
    if (k_base < block_tile_size) {
      // 计算指针偏移
      const int in_offset = smem_edge_in[k_base] * emb_dim;
      if(in_offset != last_offset) {
        cg::memcpy_async(warp, shard_in_ptr, emb_table + in_offset,
            sizeof(DATA_TYPE) * emb_dim);
        cg::wait(warp);
      }

      last_offset = in_offset;

      const int out_offset = smem_edge_out[k_base] * emb_dim;
      for (int j = lane_id; j < emb_dim; j += WARP_SIZE) {
        atomicAdd(&pooling_table[out_offset + j], shard_in_ptr[j]);
      }
    }
    k_base++;
  }
}

// 宏定义用于检查CUDA API调用的返回状态
#define CUDA_CHECK(call)                                               \
  do {                                                                 \
    hipError_t status = call;                                         \
    if (status != hipSuccess) {                                       \
      fprintf(stderr, "CUDA Error at %s:%d: %s\n", __FILE__, __LINE__, \
              hipGetErrorString(status));                             \
      exit(EXIT_FAILURE);                                              \
    }                                                                  \
  } while (0)

// ==========================================================================
// 3. Main函数：用于数据构造、内核调用和性能测试
// ==========================================================================

int main() {
  // ---- 1. 参数定义 ----
  // (此部分保持不变)

  using DataType = float;
  const int TILE_INDICES_VAL = 512;
  const int BLOCK_SIZE = 512;
  std::ifstream inFile("binary_data.bin", std::ios::binary);

  int edge_length, emb_table_length, pooling_table_length;
  int64_t emb_dim;
  inFile.read(reinterpret_cast<char *>(&edge_length), sizeof(int));
  inFile.read(reinterpret_cast<char *>(&emb_table_length), sizeof(int));
  inFile.read(reinterpret_cast<char *>(&pooling_table_length), sizeof(int));
  inFile.read(reinterpret_cast<char *>(&emb_dim), sizeof(int64_t));

  int *edge_in_cpu = reinterpret_cast<int *>(malloc(edge_length * sizeof(int)));
  int *edge_out_cpu =
      reinterpret_cast<int *>(malloc(edge_length * sizeof(int)));
  DataType *emb_table_cpu =
      reinterpret_cast<DataType *>(malloc(emb_table_length * sizeof(DataType)));
  DataType *pooling_table_cpu = reinterpret_cast<DataType *>(
      malloc(pooling_table_length * sizeof(DataType)));
  inFile.read(reinterpret_cast<char *>(edge_in_cpu), edge_length * sizeof(int));
  inFile.read(reinterpret_cast<char *>(edge_out_cpu),
              edge_length * sizeof(int));
  inFile.read(reinterpret_cast<char *>(emb_table_cpu),
              emb_table_length * sizeof(float));

  std::cout << "===== KERNEL PERFORMANCE TEST (CORRECTED) =====" << std::endl;
  std::cout << "Data Type: float" << std::endl;
  std::cout << "Embedding Dim: " << emb_dim << std::endl;
  std::cout << "Edge Length: " << edge_length << std::endl;
  std::cout << "===============================================" << std::endl;

  DataType *d_emb_table, *d_pooling_table;
  int *d_edge_in, *d_edge_out;

  int *d_visit_num;

  CUDA_CHECK(hipMalloc(&d_emb_table, emb_table_length * sizeof(DataType)));
  CUDA_CHECK(hipMalloc(&d_edge_in, edge_length * sizeof(int) + 1000));
  CUDA_CHECK(hipMalloc(&d_edge_out, edge_length * sizeof(int)));
  CUDA_CHECK(hipMalloc(&d_visit_num, pooling_table_length * sizeof(int)));

  CUDA_CHECK(
      hipMalloc(&d_pooling_table, pooling_table_length * sizeof(DataType)));

  CUDA_CHECK(hipMemcpy(d_emb_table, emb_table_cpu,
                        emb_table_length * sizeof(DataType),
                        hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_edge_in, edge_in_cpu, edge_length * sizeof(int),
                        hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_edge_out, edge_out_cpu, edge_length * sizeof(int),
                        hipMemcpyHostToDevice));
  CUDA_CHECK(
      hipMemset(d_pooling_table, 0, pooling_table_length * sizeof(DataType)));
  CUDA_CHECK(
      hipMemset(d_visit_num, 0, pooling_table_length * sizeof(int)));

  const dim3 blockDim(BLOCK_SIZE);
  const dim3 gridDim(ITER(edge_length, TILE_INDICES_VAL));
  std::cout << "Grid Dim: " << gridDim.x << ", Block Dim: " << blockDim.x
            << std::endl;
  size_t smem_size = 2 * TILE_INDICES_VAL * sizeof(int) +
                     (BLOCK_SIZE / 32 * emb_dim) * sizeof(DataType);

  hipEvent_t start, stop;
  CUDA_CHECK(hipEventCreate(&start));
  CUDA_CHECK(hipEventCreate(&stop));

  mark_counts_cas_kernel<<<gridDim, blockDim>>>(d_edge_out, d_visit_num,
    edge_length);
  CUDA_CHECK(hipDeviceSynchronize());
  gpu_pooling_forward_async_kernel<DataType, TILE_INDICES_VAL, BLOCK_SIZE>
      <<<gridDim, blockDim, smem_size>>>(d_emb_table, d_edge_in, d_edge_out,
                                         d_pooling_table, emb_dim, edge_length,d_visit_num);

  CUDA_CHECK(hipDeviceSynchronize());

  // int num_runs = 100;

  // CUDA_CHECK(hipEventRecord(start));
  // for (int i = 0; i < num_runs; ++i) {
  //   gpu_pooling_forward_async_kernel<DataType, TILE_INDICES_VAL, BLOCK_SIZE>
  //       <<<gridDim, blockDim, smem_size>>>(d_emb_table, d_edge_in, d_edge_out,
  //                                          d_pooling_table, emb_dim,
  //                                          edge_length, d_visit_num);
  // }
  // CUDA_CHECK(hipEventRecord(stop));

  // CUDA_CHECK(hipEventSynchronize(stop));
  // float total_time = 0;
  // CUDA_CHECK(hipEventElapsedTime(&total_time, start, stop));

  // float average_time_ms = total_time / num_runs;
  // std::cout << "\n--- Performance Results ---" << std::endl;
  // std::cout << "Number of test runs: " << num_runs << std::endl;
  // std::cout << "Average kernel execution time: " << average_time_ms << " ms"
  //           << std::endl;

  // std::cout << "\n--- Verification ---" << std::endl;

  // std::cout
  //     << "Resetting GPU buffer and running kernel once for verification..."
  //     << std::endl;
  // CUDA_CHECK(
  //     hipMemset(d_pooling_table, 0, pooling_table_length * sizeof(DataType)));

  // gpu_pooling_forward_async_kernel<DataType, TILE_INDICES_VAL, BLOCK_SIZE>
  //     <<<gridDim, blockDim, smem_size>>>(d_emb_table, d_edge_in, d_edge_out,
  //                                        d_pooling_table, emb_dim, edge_length, d_visit_num);
  // CUDA_CHECK(hipDeviceSynchronize());  // 确保内核执行完毕

  // std::vector<DataType> h_gpu_result(pooling_table_length);
  // CUDA_CHECK(hipMemcpy(h_gpu_result.data(), d_pooling_table,
  //                       h_gpu_result.size() * sizeof(DataType),
  //                       hipMemcpyDeviceToHost));
  // memset(pooling_table_cpu, 0, pooling_table_length * sizeof(DataType));

  // std::cout << "Calculating reference result on CPU..." << std::endl;
  // for (int i = 0; i < edge_length; ++i) {
  //   int in_node = edge_in_cpu[i];
  //   int out_node = edge_out_cpu[i];
  //   for (int64_t d = 0; d < emb_dim; ++d) {
  //     pooling_table_cpu[out_node * emb_dim + d] +=
  //         emb_table_cpu[in_node * emb_dim + d];
  //   }
  // }

  // double total_absolute_error = 0.0;
  // for (size_t i = 0; i < pooling_table_length; ++i) {
  //   total_absolute_error += std::abs(pooling_table_cpu[i] - h_gpu_result[i]);
  // }

  // std::cout << "Total absolute error between CPU and GPU: "
  //           << total_absolute_error << std::endl;
  // if (total_absolute_error < 1e-1) {
  //   std::cout << "Result verification PASSED." << std::endl;
  // } else {
  //   std::cout << "Result verification FAILED." << std::endl;
  // }

  CUDA_CHECK(hipEventDestroy(start));
  CUDA_CHECK(hipEventDestroy(stop));
  CUDA_CHECK(hipFree(d_emb_table));
  CUDA_CHECK(hipFree(d_edge_in));
  CUDA_CHECK(hipFree(d_edge_out));
  CUDA_CHECK(hipFree(d_pooling_table));

  return 0;
}
