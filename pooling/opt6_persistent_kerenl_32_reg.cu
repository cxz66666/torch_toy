#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <random>
#include <chrono>
#include <hip/hip_runtime_api.h>
#include <fstream>
#include <cstdint>
#include <hip/hip_bf16.h>
#include <hip/hip_fp16.h>
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/memcpy_async.h>

// ==========================================================================
// 1. 缺失的定义补充
// ==========================================================================

/**
 * @brief 计算向上取整的整数除法 (ceil(a/b))。
 * @param a 分子
 * @param b 分母
 */
#define ITER(x, y) (x + y - 1) / y

template <typename DATA_TYPE,
          typename std::enable_if_t<
              !std::is_same<__half, DATA_TYPE>::value &&
              !std::is_same<__hip_bfloat16, DATA_TYPE>::value> * = nullptr>
__device__ __forceinline__ void fastSpecializedAtomicAdd(DATA_TYPE *base,
                                                         int64_t offset,
                                                         const int64_t length,
                                                         DATA_TYPE value) {
  atomicAdd(base + offset, value);
}

template <class DATA_TYPE>
__device__ __forceinline__ void fastAtomicAdd(DATA_TYPE *base, int64_t offset,
                                              const int64_t length,
                                              DATA_TYPE value) {
  fastSpecializedAtomicAdd(base, offset, length, value);
}

// struct SharedMemory {
//   int edge_in_tile[TILE_K_PER_BLOCK];
//   int edge_out_tile[TILE_K_PER_BLOCK];
//   // 双缓冲，用于隐藏内存延迟
//   // 布局: [buffer_idx][warp_idx][element_idx]
//   DATA_TYPE emb_buffer[2][NUM_WARPS * EMB_DIM_STATIC];
// };

template <typename DATA_TYPE, int TILE_K_PER_BLOCK, int BLOCK_THREADS>
__global__ void __launch_bounds__(BLOCK_THREADS)
    gpu_pooling_forward_async_kernel(
        const DATA_TYPE *__restrict__ emb_table,
        const int *__restrict__ edge_in, const int *__restrict__ edge_out,
        DATA_TYPE *__restrict__ pooling_table, const int64_t emb_dim,
        const int edge_length,
        int *work_tile_idx /* 新增：全局工作块原子计数器 */) {
  constexpr int WARP_SIZE = 32;
  constexpr int NUM_WARPS = BLOCK_THREADS / WARP_SIZE;
  const int thread_id = threadIdx.x;
  const int warp_id = thread_id / WARP_SIZE;
  const int lane_id = thread_id % WARP_SIZE;

  // 动态共享内存布局
  extern __shared__ char smem_storage[];
  int *smem_edge_in = reinterpret_cast<int *>(smem_storage);
  int *smem_edge_out = smem_edge_in + TILE_K_PER_BLOCK;
  int *smem_work_tile_idx =
      reinterpret_cast<int *>(smem_edge_out + TILE_K_PER_BLOCK);
  while (true) {
    // 1. 原子地获取一个工作块索引 (work-stealing)
    if (thread_id == 0) {
      *smem_work_tile_idx = atomicAdd(work_tile_idx, 1);
    }
    __syncthreads();
    const int tile_idx = *smem_work_tile_idx;
    // 2. 根据领取的索引计算该块处理的边范围
    const int block_tile_start = tile_idx * TILE_K_PER_BLOCK;

    // 3. 检查是否所有工作都已完成，若是则退出循环
    if (block_tile_start >= edge_length) {
      break;
    }

    const int block_tile_end =
        min(block_tile_start + TILE_K_PER_BLOCK, edge_length);
    const int block_tile_size = block_tile_end - block_tile_start;

    // 预加载边索引到共享内存
    __syncthreads();  // 确保上一轮迭代的共享内存使用已完成
    for (int i = thread_id; i < block_tile_size; i += BLOCK_THREADS) {
      smem_edge_in[i] = edge_in[block_tile_start + i];
      smem_edge_out[i] = edge_out[block_tile_start + i];
    }
    int k_base = 0;
    __syncthreads();

    // 主处理循环 (处理一个tile内部的边)
    while (true) {
      if (k_base >= block_tile_size) break;

      // 每个warp处理一个边
      const int k_warp = k_base + warp_id;
      if (k_warp < block_tile_size) {
        const int in_offset = smem_edge_in[k_warp] * emb_dim;
        const int out_offset = smem_edge_out[k_warp] * emb_dim;

        // 处理embedding维度
        for (int j = lane_id; j < emb_dim; j += WARP_SIZE) {
          atomicAdd(&pooling_table[out_offset + j], emb_table[in_offset + j]);
        }
      }
      k_base += NUM_WARPS;
    }
  }  // 结束主 while 循环
}

// 宏定义用于检查CUDA API调用的返回状态
#define CUDA_CHECK(call)                                               \
  do {                                                                 \
    hipError_t status = call;                                         \
    if (status != hipSuccess) {                                       \
      fprintf(stderr, "CUDA Error at %s:%d: %s\n", __FILE__, __LINE__, \
              hipGetErrorString(status));                             \
      exit(EXIT_FAILURE);                                              \
    }                                                                  \
  } while (0)

// ==========================================================================
// 3. Main函数：用于数据构造、内核调用和性能测试
// ==========================================================================

int main() {
  // ---- 1. 参数定义 ----
  // (此部分保持不变)

  using DataType = float;
  const int TILE_INDICES_VAL = 512;
  const int BLOCK_SIZE = 512;
  std::ifstream inFile("binary_data.bin", std::ios::binary);

  int edge_length, emb_table_length, pooling_table_length;
  int64_t emb_dim;
  inFile.read(reinterpret_cast<char *>(&edge_length), sizeof(int));
  inFile.read(reinterpret_cast<char *>(&emb_table_length), sizeof(int));
  inFile.read(reinterpret_cast<char *>(&pooling_table_length), sizeof(int));
  inFile.read(reinterpret_cast<char *>(&emb_dim), sizeof(int64_t));

  int *edge_in_cpu = reinterpret_cast<int *>(malloc(edge_length * sizeof(int)));
  int *edge_out_cpu =
      reinterpret_cast<int *>(malloc(edge_length * sizeof(int)));
  DataType *emb_table_cpu =
      reinterpret_cast<DataType *>(malloc(emb_table_length * sizeof(DataType)));
  DataType *pooling_table_cpu = reinterpret_cast<DataType *>(
      malloc(pooling_table_length * sizeof(DataType)));
  inFile.read(reinterpret_cast<char *>(edge_in_cpu), edge_length * sizeof(int));
  inFile.read(reinterpret_cast<char *>(edge_out_cpu),
              edge_length * sizeof(int));
  inFile.read(reinterpret_cast<char *>(emb_table_cpu),
              emb_table_length * sizeof(float));

  std::cout << "===== KERNEL PERFORMANCE TEST (CORRECTED) =====" << std::endl;
  std::cout << "Data Type: float" << std::endl;
  std::cout << "Embedding Dim: " << emb_dim << std::endl;
  std::cout << "Edge Length: " << edge_length << std::endl;
  std::cout << "===============================================" << std::endl;

  DataType *d_emb_table, *d_pooling_table;
  int *d_edge_in, *d_edge_out;

  CUDA_CHECK(hipMalloc(&d_emb_table, emb_table_length * sizeof(DataType)));
  CUDA_CHECK(hipMalloc(&d_edge_in, edge_length * sizeof(int)));
  CUDA_CHECK(hipMalloc(&d_edge_out, edge_length * sizeof(int)));
  CUDA_CHECK(
      hipMalloc(&d_pooling_table, pooling_table_length * sizeof(DataType)));

  CUDA_CHECK(hipMemcpy(d_emb_table, emb_table_cpu,
                        emb_table_length * sizeof(DataType),
                        hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_edge_in, edge_in_cpu, edge_length * sizeof(int),
                        hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_edge_out, edge_out_cpu, edge_length * sizeof(int),
                        hipMemcpyHostToDevice));
  CUDA_CHECK(
      hipMemset(d_pooling_table, 0, pooling_table_length * sizeof(DataType)));

  const dim3 blockDim(BLOCK_SIZE);
  const dim3 gridDim(78 * 4);
  std::cout << "Grid Dim: " << gridDim.x << ", Block Dim: " << blockDim.x
            << std::endl;
  size_t smem_size = 2 * TILE_INDICES_VAL * sizeof(int) + 1 * sizeof(int);

  int *d_work_tile_idx;
  hipMalloc(&d_work_tile_idx, sizeof(int));
  hipMemset(d_work_tile_idx, 0, sizeof(int));
  hipEvent_t start, stop;
  CUDA_CHECK(hipEventCreate(&start));
  CUDA_CHECK(hipEventCreate(&stop));

  gpu_pooling_forward_async_kernel<DataType, TILE_INDICES_VAL, BLOCK_SIZE>
      <<<gridDim, blockDim, smem_size>>>(d_emb_table, d_edge_in, d_edge_out,
                                         d_pooling_table, emb_dim, edge_length,
                                         d_work_tile_idx);

  CUDA_CHECK(hipDeviceSynchronize());

  // int num_runs = 100;

  // CUDA_CHECK(hipEventRecord(start));
  // for (int i = 0; i < num_runs; ++i) {
  //   hipMemset(d_work_tile_idx, 0, sizeof(int));
  //   gpu_pooling_forward_async_kernel<DataType, TILE_INDICES_VAL, BLOCK_SIZE>
  //       <<<gridDim, blockDim, smem_size>>>(d_emb_table, d_edge_in,
  //       d_edge_out,
  //                                          d_pooling_table, emb_dim,
  //                                          edge_length, d_work_tile_idx);
  // }
  // CUDA_CHECK(hipEventRecord(stop));

  // CUDA_CHECK(hipEventSynchronize(stop));
  // float total_time = 0;
  // CUDA_CHECK(hipEventElapsedTime(&total_time, start, stop));

  // float average_time_ms = total_time / num_runs;
  // std::cout << "\n--- Performance Results ---" << std::endl;
  // std::cout << "Number of test runs: " << num_runs << std::endl;
  // std::cout << "Average kernel execution time: " << average_time_ms << " ms"
  //           << std::endl;

  // std::cout << "\n--- Verification ---" << std::endl;

  // std::cout
  //     << "Resetting GPU buffer and running kernel once for verification..."
  //     << std::endl;
  // CUDA_CHECK(
  //     hipMemset(d_pooling_table, 0, pooling_table_length *
  //     sizeof(DataType)));

  // hipMemset(d_work_tile_idx, 0, sizeof(int));
  // gpu_pooling_forward_async_kernel<DataType, TILE_INDICES_VAL, BLOCK_SIZE>
  //     <<<gridDim, blockDim, smem_size>>>(d_emb_table, d_edge_in, d_edge_out,
  //                                        d_pooling_table, emb_dim,
  //                                        edge_length, d_work_tile_idx);
  // CUDA_CHECK(hipDeviceSynchronize());  // 确保内核执行完毕

  // std::vector<DataType> h_gpu_result(pooling_table_length);
  // CUDA_CHECK(hipMemcpy(h_gpu_result.data(), d_pooling_table,
  //                       h_gpu_result.size() * sizeof(DataType),
  //                       hipMemcpyDeviceToHost));
  // memset(pooling_table_cpu, 0, pooling_table_length * sizeof(DataType));

  // std::cout << "Calculating reference result on CPU..." << std::endl;
  // for (int i = 0; i < edge_length; ++i) {
  //   int in_node = edge_in_cpu[i];
  //   int out_node = edge_out_cpu[i];
  //   for (int64_t d = 0; d < emb_dim; ++d) {
  //     pooling_table_cpu[out_node * emb_dim + d] +=
  //         emb_table_cpu[in_node * emb_dim + d];
  //   }
  // }

  // double total_absolute_error = 0.0;
  // for (size_t i = 0; i < pooling_table_length; ++i) {
  //   total_absolute_error += std::abs(pooling_table_cpu[i] - h_gpu_result[i]);
  // }

  // std::cout << "Total absolute error between CPU and GPU: "
  //           << total_absolute_error << std::endl;
  // if (total_absolute_error < 1e-1) {
  //   std::cout << "Result verification PASSED." << std::endl;
  // } else {
  //   std::cout << "Result verification FAILED." << std::endl;
  // }

  CUDA_CHECK(hipEventDestroy(start));
  CUDA_CHECK(hipEventDestroy(stop));
  CUDA_CHECK(hipFree(d_emb_table));
  CUDA_CHECK(hipFree(d_edge_in));
  CUDA_CHECK(hipFree(d_edge_out));
  CUDA_CHECK(hipFree(d_pooling_table));

  return 0;
}
