#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <random>
#include <chrono>
#include <hip/hip_runtime_api.h>
#include <fstream>
#include <cstdint>
#include <hip/hip_bf16.h>
#include <hip/hip_fp16.h>
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/memcpy_async.h>

// /usr/local/cuda/bin/nvcc  -gencode arch=compute_90a,code=sm_90a -lcudart
// opt1.cu

// ==========================================================================
// 1. 缺失的定义补充
// ==========================================================================

/**
 * @brief 计算向上取整的整数除法 (ceil(a/b))。
 * @param a 分子
 * @param b 分母
 */
#define ITER(x, y) (x + y - 1) / y

template <typename DATA_TYPE,
          typename std::enable_if_t<
              !std::is_same<__half, DATA_TYPE>::value &&
              !std::is_same<__hip_bfloat16, DATA_TYPE>::value> * = nullptr>
__device__ __forceinline__ void fastSpecializedAtomicAdd(DATA_TYPE *base,
                                                         int64_t offset,
                                                         const int64_t length,
                                                         DATA_TYPE value) {
  atomicAdd(base + offset, value);
}

template <class DATA_TYPE>
__device__ __forceinline__ void fastAtomicAdd(DATA_TYPE *base, int64_t offset,
                                              const int64_t length,
                                              DATA_TYPE value) {
  fastSpecializedAtomicAdd(base, offset, length, value);
}

// struct SharedMemory {
//   int edge_in_tile[TILE_K_PER_BLOCK];
//   int edge_out_tile[TILE_K_PER_BLOCK];
//   // 双缓冲，用于隐藏内存延迟
//   // 布局: [buffer_idx][warp_idx][element_idx]
//   DATA_TYPE emb_buffer[2][NUM_WARPS * EMB_DIM_STATIC];
// };

#define BLOCK_READ_EMB_ 32
// #define EMB_DIM_MAX_ 1024
#define TILE_INDICES_ 16

template <typename DATA_TYPE, int BLOCK_READ_EMB, int TILE_INDICES>
__global__ void __launch_bounds__(512, 2)
    gpu_pooling_forward_kernel(const DATA_TYPE *__restrict__ emb_table,
                               const int *__restrict__ edge_in,
                               const int *__restrict__ edge_out,
                               DATA_TYPE *__restrict__ pooling_table,
                               const int64_t emb_dim, const int edge_length) {
  // the thread block size used to read indices in a tile: (block_read_indices,
  // BLOCK_READ_EMB)
  const int block_read_indices = blockDim.x / BLOCK_READ_EMB;
  // In row_ids/indice_values array, for each block, traverse times.
  const int iter_indices_block = ITER(edge_length, TILE_INDICES);
  // In one tile, for each thread, traverse times.
  const int iter_indices_thread = ITER(TILE_INDICES, block_read_indices);
  // In one emb, for each thread, traverse times.
  const int64_t iter_emb = ITER(emb_dim, BLOCK_READ_EMB);

  int64_t indice_value = 0;
  int64_t row_id = 0;

#pragma unroll
  for (int b = 0; b < iter_indices_block; b++) {
    const int block_offset_indices =
        (b * gridDim.x + blockIdx.x) * TILE_INDICES;
    if (block_offset_indices >= edge_length) {
      return;
    }

    const int end = min(block_offset_indices + TILE_INDICES, edge_length);

#pragma unroll
    for (int i = 0; i < iter_indices_thread; i++) {
      const int thread_idx_indices =
          i * block_read_indices + threadIdx.x / BLOCK_READ_EMB;
      const int indice_idx = block_offset_indices + thread_idx_indices;
      if (indice_idx >= end) break;
      indice_value = edge_in[indice_idx] * emb_dim;
      row_id = edge_out[indice_idx] * emb_dim;
#pragma unroll
      for (int64_t j = 0; j < iter_emb; j++) {
        const int64_t thread_idx_emb =
            j * BLOCK_READ_EMB + threadIdx.x % BLOCK_READ_EMB;
        if (thread_idx_emb >= emb_dim) break;
        const int64_t emb_idx = indice_value + thread_idx_emb;
        // const int pooling_idx = row_id + thread_idx_emb;
        fastAtomicAdd(reinterpret_cast<DATA_TYPE *>(pooling_table + row_id),
                      thread_idx_emb, emb_dim,
                      static_cast<DATA_TYPE>(emb_table[emb_idx]));
      }
    }
  }
}

// 宏定义用于检查CUDA API调用的返回状态
#define CUDA_CHECK(call)                                               \
  do {                                                                 \
    hipError_t status = call;                                         \
    if (status != hipSuccess) {                                       \
      fprintf(stderr, "CUDA Error at %s:%d: %s\n", __FILE__, __LINE__, \
              hipGetErrorString(status));                             \
      exit(EXIT_FAILURE);                                              \
    }                                                                  \
  } while (0)

// ==========================================================================
// 3. Main函数：用于数据构造、内核调用和性能测试
// ==========================================================================

int main() {
  // ---- 1. 参数定义 ----
  // (此部分保持不变)

  using DataType = float;
  std::ifstream inFile("binary_data.bin", std::ios::binary);

  int edge_length, emb_table_length, pooling_table_length;
  int64_t emb_dim;
  inFile.read(reinterpret_cast<char *>(&edge_length), sizeof(int));
  inFile.read(reinterpret_cast<char *>(&emb_table_length), sizeof(int));
  inFile.read(reinterpret_cast<char *>(&pooling_table_length), sizeof(int));
  inFile.read(reinterpret_cast<char *>(&emb_dim), sizeof(int64_t));

  int *edge_in_cpu = reinterpret_cast<int *>(malloc(edge_length * sizeof(int)));
  int *edge_out_cpu =
      reinterpret_cast<int *>(malloc(edge_length * sizeof(int)));
  DataType *emb_table_cpu =
      reinterpret_cast<DataType *>(malloc(emb_table_length * sizeof(DataType)));
  DataType *pooling_table_cpu = reinterpret_cast<DataType *>(
      malloc(pooling_table_length * sizeof(DataType)));
  inFile.read(reinterpret_cast<char *>(edge_in_cpu), edge_length * sizeof(int));
  inFile.read(reinterpret_cast<char *>(edge_out_cpu),
              edge_length * sizeof(int));
  inFile.read(reinterpret_cast<char *>(emb_table_cpu),
              emb_table_length * sizeof(float));

  std::cout << "===== KERNEL PERFORMANCE TEST (CORRECTED) =====" << std::endl;
  std::cout << "Data Type: float" << std::endl;
  std::cout << "Embedding Dim: " << emb_dim << std::endl;
  std::cout << "Edge Length: " << edge_length << std::endl;
  std::cout << "===============================================" << std::endl;

  // ---- 3. 设备端(GPU)内存分配 ----
  // (此部分保持不变)
  DataType *d_emb_table, *d_pooling_table;
  int *d_edge_in, *d_edge_out;

  CUDA_CHECK(hipMalloc(&d_emb_table, emb_table_length * sizeof(DataType)));
  CUDA_CHECK(hipMalloc(&d_edge_in, edge_length * sizeof(int)));
  CUDA_CHECK(hipMalloc(&d_edge_out, edge_length * sizeof(int)));
  CUDA_CHECK(
      hipMalloc(&d_pooling_table, pooling_table_length * sizeof(DataType)));

  // ---- 4. 数据从主机到设备 ----
  // (此部分保持不变)
  CUDA_CHECK(hipMemcpy(d_emb_table, emb_table_cpu,
                        emb_table_length * sizeof(DataType),
                        hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_edge_in, edge_in_cpu, edge_length * sizeof(int),
                        hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_edge_out, edge_out_cpu, edge_length * sizeof(int),
                        hipMemcpyHostToDevice));
  CUDA_CHECK(
      hipMemset(d_pooling_table, 0, pooling_table_length * sizeof(DataType)));
  // ---- 5. 内核启动配置 ----
  // (此部分保持不变)
  const dim3 blockDim(512);
  const dim3 gridDim(4096);
  std::cout << "Grid Dim: " << gridDim.x << ", Block Dim: " << blockDim.x
            << std::endl;
  size_t smem_size = 0;
  // ---- 6. 性能测试 ----
  hipEvent_t start, stop;
  CUDA_CHECK(hipEventCreate(&start));
  CUDA_CHECK(hipEventCreate(&stop));

  gpu_pooling_forward_kernel<DataType, BLOCK_READ_EMB_, TILE_INDICES_>
      <<<gridDim, blockDim, smem_size>>>(d_emb_table, d_edge_in, d_edge_out,
                                         d_pooling_table, emb_dim, edge_length);

  CUDA_CHECK(hipDeviceSynchronize());

//   int num_runs = 100;

//   // 开始正式计时
//   CUDA_CHECK(hipEventRecord(start));
//   for (int i = 0; i < num_runs; ++i) {
//     // 在性能测试中，通常我们不把内存清零的时间算进去，假设输入buffer是准备好的
//     // 如果需要包含清零时间，则应将hipMemset也放入循环
//     gpu_pooling_forward_kernel<DataType, BLOCK_READ_EMB_, TILE_INDICES_>
//         <<<gridDim, blockDim, smem_size>>>(d_emb_table, d_edge_in, d_edge_out,
//                                            d_pooling_table, emb_dim,
//                                            edge_length);
//   }
//   CUDA_CHECK(hipEventRecord(stop));

//   CUDA_CHECK(hipEventSynchronize(stop));
//   float total_time = 0;
//   CUDA_CHECK(hipEventElapsedTime(&total_time, start, stop));

//   float average_time_ms = total_time / num_runs;
//   std::cout << "\n--- Performance Results ---" << std::endl;
//   std::cout << "Number of test runs: " << num_runs << std::endl;
//   std::cout << "Average kernel execution time: " << average_time_ms << " ms"
//             << std::endl;

//   // ---- 7. 结果验证 ----
//   std::cout << "\n--- Verification ---" << std::endl;

//   std::cout
//       << "Resetting GPU buffer and running kernel once for verification..."
//       << std::endl;
//   CUDA_CHECK(
//       hipMemset(d_pooling_table, 0, pooling_table_length * sizeof(DataType)));

//   // 在干净的缓冲上**只运行一次**内核以获取正确结果
//   gpu_pooling_forward_kernel<DataType, BLOCK_READ_EMB_, TILE_INDICES_>
//       <<<gridDim, blockDim, smem_size>>>(d_emb_table, d_edge_in, d_edge_out,
//                                          d_pooling_table, emb_dim, edge_length);
//   CUDA_CHECK(hipDeviceSynchronize());  // 确保内核执行完毕

//   // 将单次运行的GPU结果拷贝回CPU
//   std::vector<DataType> h_gpu_result(pooling_table_length);
//   CUDA_CHECK(hipMemcpy(h_gpu_result.data(), d_pooling_table,
//                         h_gpu_result.size() * sizeof(DataType),
//                         hipMemcpyDeviceToHost));
//   memset(pooling_table_cpu, 0, pooling_table_length * sizeof(DataType));
//   // 在CPU上执行相同的操作以获得参照结果
//   std::cout << "Calculating reference result on CPU..." << std::endl;
//   for (int i = 0; i < edge_length; ++i) {
//     int in_node = edge_in_cpu[i];
//     int out_node = edge_out_cpu[i];
//     for (int64_t d = 0; d < emb_dim; ++d) {
//       pooling_table_cpu[out_node * emb_dim + d] +=
//           emb_table_cpu[in_node * emb_dim + d];
//     }
//   }

//   // 比较CPU和GPU的结果
//   double total_absolute_error = 0.0;
//   for (size_t i = 0; i < pooling_table_length; ++i) {
//     total_absolute_error += std::abs(pooling_table_cpu[i] - h_gpu_result[i]);
//   }

//   std::cout << "Total absolute error between CPU and GPU: "
//             << total_absolute_error << std::endl;
//   if (total_absolute_error < 1e-1) {  // 容忍微小的浮点误差
//     std::cout << "Result verification PASSED." << std::endl;
//   } else {
//     std::cout << "Result verification FAILED." << std::endl;
//   }

  // ---- 8. 资源清理 ----
  // (此部分保持不变)
  CUDA_CHECK(hipEventDestroy(start));
  CUDA_CHECK(hipEventDestroy(stop));
  CUDA_CHECK(hipFree(d_emb_table));
  CUDA_CHECK(hipFree(d_edge_in));
  CUDA_CHECK(hipFree(d_edge_out));
  CUDA_CHECK(hipFree(d_pooling_table));

  return 0;
}
